#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : where_is_waldo.cu
 Author      : volk, hettich
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include "where_is_waldo.h"
#include "PPM.hh"

#define MAX_THREADS 512

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

# define M_PI           3.14159265358979323846  /* pi */

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
/*__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}*/

__global__ void gaussKernel(float *_src, float *_dst, float* _weight, int _width, int _w, int _h) {
    int _x = blockIdx.x * blockDim.x + threadIdx.x;
    int _y = blockIdx.y * blockDim.y + threadIdx.y;
    //int pos = y * _w + x;

    if (_x >= _w || _y >= _h) {
        return;
    }

    int half = _width/2;
    float blur = 0.f;

    for (int y = -half; y <= half; y++) {
        for (int x = -half; x <= half; x++) {
            int w = min(max(_x+x, 0), _w);
            int h = min(max(_y+y, 0), _h);

            int idx = w + _w*h;

            float pixel = _src[idx];

            idx = (y+half)*_width + x+half;

            float weight = _weight[idx];

            blur += pixel*weight;
        }
    }

    _dst[_x + _y*_w] = blur;
}

/**
 * Host function that copies the data and launches the work on GPU
 */
/*float *gpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));
	
	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

float *cpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
	return rc;
}


void initialize(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = .5*(i+1);
}*/

int run(char* imagePath, char* outputPath)
{
    /** BEGIN init kernel. */
    float* img;

    int w, h;
    // read imagePath (input image).
    ppm::readPPM(imagePath, w, h, &img);

    int nPix = w*h;
    float* gpuImg;
    float* gpuResult;

    hipMalloc((void**) &gpuImg, nPix*3*sizeof(float));
    hipMalloc((void**) &gpuResult, nPix*3*sizeof(float));

    hipMemcpy(gpuImg, img, nPix*3*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(gpuResult, 0, nPix*3*sizeof(float));
    /** END init kernel. */

    /** BEGIN create gaussian */
    const int width = 9;
    const float sigma = 2.f;

    const int half = width/2;
    float sum = 0.f;

    /*vector<float> matrix;
    matrix.resize(width*width);*/
    float* matrix;

    for (int y = -half; y <= half; y++) {
        for (int x = -half; x <= half; x++) {
            float weight = std::exp(-static_cast<float>(x*x+y*y)/(2.f*sigma*sigma));
            int idx = (y+half)*width + x+half;

            matrix[idx] = weight;
            sum += weight;
        }
    }

    float normal = 1.f/sum;

    for (int y = -half; y <= half; y++) {
        for (int x = -half; x <= half; x++) {
            int idx = (y+half)*width + x+half;

            matrix[idx] *= normal;
        }
    }
    /** END create gaussian */

    /** BEGIN run kernel. */
    static const int BLOCK_WIDTH = 32;
    int x = w/BLOCK_WIDTH;
    int y = h/BLOCK_WIDTH;

    const dim3 grid (x, y, 1);
    const dim3 block (BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // run gaussian blur kernel.
    std::cout<<"would execute kernerl."<<std::endl;
    gaussKernel<<< grid, block >>>(gpuImg, gpuResult, matrix, width, w, h);
    /** END run kernel. */

    /** BEGIN save to disk */
    hipMemcpy(img, gpuResult, nPix*3*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(gpuResult);
    hipFree(gpuImg);

    ppm::writePPM(outputPath, w, h, (float*) img);

    delete[] img;
    /** BEGIN save to disk */

    std::cout<<"done"<<std::endl;

    return 0;

    /*static const int WORK_SIZE = 65530;
	float *data = new float[WORK_SIZE];

	initialize (data, WORK_SIZE);

	float *recCpu = cpuReciprocal(data, WORK_SIZE);
	float *recGpu = gpuReciprocal(data, WORK_SIZE);
	float cpuSum = std::accumulate (recCpu, recCpu+WORK_SIZE, 0.0);
	float gpuSum = std::accumulate (recGpu, recGpu+WORK_SIZE, 0.0);

    // Verify the results
	std::cout<<"gpuSum2 = "<<gpuSum<< " cpuSum = " <<cpuSum<<std::endl;

    // Free memory
	delete[] data;
	delete[] recCpu;
	delete[] recGpu;

    return 0;*/
}

/*
int main(int argc, char *argv[])
{
	string imagePath;
	string outputPath;
	/*
	// load image to perform gaussian blur.
		if (argc > 1) {
			imagePath = string(argv[1]);
			if (argc > 2) {
				outputPath = string(argv[2]);
			} else {
				outputPath = "output.jpg";
			}
		} else {
			cerr << "Please provide an input (and output) image path as arguments to this application." << endl;
			exit(1);
		}
	return run(imagePath, outputPath);
	run(imagePath,outputPath);
}
*/


/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

