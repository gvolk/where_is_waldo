#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : gpu_classifier.cu
 Author      : volk, hettich
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA logistic regression classifier
 ============================================================================
 */
#include "gpu_classifier.h"
#define THREADS_PER_BLOCK 512


#define MAX_THREADS 256

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


__global__ void trainKernel(int* labels, float* features, int num_features, double *beta, double* gputmpbeta)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    double z, proby;
    int pos;

    if (x  >= num_features) {
        return;
    }

    pos = x * FEAT_LEN ;

    for(int j = 0; j < FEAT_LEN; j++) {
            z += beta[j] * (double)features[pos + j];
    }

    proby = (1.0 / (1.0 + exp(-z)));

    for(int j = 0; j < FEAT_LEN; j++) {
        //save #num_features betas of the first feature then #num_features beats for the second for later reduce them
        gputmpbeta[j * num_features + x] += (double) LEARN_CONST * ((double)features[pos + j]*(labels[x] - proby));
    }

}

//reduce betas of one feature previously computed by trainKernel
__global__ void reduceBetas(int num_features, int numBlocks, int feature_beta_idx, double* gpuResultBeta1, double* gpuResultBeta2, double* gpubeta)
{
    extern __shared__ double partialSum[];

    int x = blockIdx.x * blockDim.x  + threadIdx.x;

    int beta_idx = feature_beta_idx * num_features + x;

    unsigned int t = threadIdx.x;

    if (x  >= num_features) {
        partialSum[t] = 0;
        return;
    }

    partialSum[t] = gpuResultBeta1[beta_idx];


    for (unsigned int stride = blockDim.x / 2 ; stride > 1; stride >>= 1) {
        __syncthreads();
        if (t < stride) {
            partialSum[t] += partialSum[t+stride];
        }
    }

    __syncthreads();

    //if numBlocks = 1 then save to gpubeta for next epoch
    if (t==0){
        if (numBlocks == 1)
        {
            gpubeta[feature_beta_idx * numBlocks + blockIdx.x] = partialSum[0] + partialSum[1];
        }
        else
        {
            gpuResultBeta2[feature_beta_idx * numBlocks + blockIdx.x] = partialSum[0] + partialSum[1];
        }
    }
}


//label and features are input variables, beta is the output variable
int train_gpu(int* labels, float* features, int num_features, double *beta)
{
    std::pair<float,float> p_correct, new_correct;
    int* cpulabels;
    double* beta_array;
    std::pair<float,float> * p_corr;
    float* gpufeatures;
    int* gpulabels;
    double* gpuResultBeta1;
    double* gpuResultBeta2;
    double* gpuResultBeta3;
    double* gpubeta;
    int i, epochs;

    unsigned int numBlocks = num_features / THREADS_PER_BLOCK +1;
    unsigned int numBlocks2 = numBlocks / THREADS_PER_BLOCK +1;

    std::cout << "numBlocks" << numBlocks;

    hipHostMalloc((void**) &cpulabels, num_features * sizeof(int));
    hipHostMalloc((void**) &beta_array, EPOCHS * FEAT_LEN * sizeof(double));
    hipHostMalloc((void**) &p_corr, EPOCHS * FEAT_LEN * sizeof(std::pair<float,float>));
    for(i = 0; i < num_features; i++)
    {
        cpulabels[i]=0;
    }

    hipMalloc((void**) &gpufeatures, num_features * FEAT_LEN * sizeof(float));
    hipMemcpy(gpufeatures, features, num_features * FEAT_LEN * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**) &gpulabels, num_features * sizeof(int));
    hipMemcpy(gpulabels, labels,   num_features * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**) &gpubeta, FEAT_LEN * sizeof(double));
    hipMemcpy(gpubeta, beta,     FEAT_LEN * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void**) &gpuResultBeta1, num_features * FEAT_LEN * sizeof(double));
    hipMemset(gpuResultBeta1, 0,        num_features * FEAT_LEN * sizeof(double));

    hipMalloc((void**) &gpuResultBeta2, numBlocks * FEAT_LEN * sizeof(double));
    hipMemset(gpuResultBeta2, 0,        numBlocks * FEAT_LEN * sizeof(double));

    hipMalloc((void**) &gpuResultBeta3, numBlocks2 * FEAT_LEN * sizeof(double));
    hipMemset(gpuResultBeta3, 0,        numBlocks2 * FEAT_LEN * sizeof(double));

    for(epochs = 0; epochs < EPOCHS; epochs++)
    {
        // train one round to get all partial betas
        trainKernel<<< numBlocks, THREADS_PER_BLOCK >>>(gpulabels, gpufeatures, num_features, gpubeta, gpuResultBeta1);


        // we need three reduce because otherwise there are too much threads per block because we have too much features
        //furthermore 3 tmp results on gpu are necessary because gpuResultBeta1 is needed in the next iteration and must not change
        // reduce all 9 betas for each feature
        for(i= 0; i < FEAT_LEN; i++)
        {
            reduceBetas<<< numBlocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double) >>>(num_features, numBlocks, i, gpuResultBeta1, gpuResultBeta2, gpubeta);
        }

        // reduce all 9 betas for each feature
        for(i= 0; i < FEAT_LEN; i++)
        {
            reduceBetas<<< numBlocks2, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double) >>>(numBlocks, numBlocks2, i, gpuResultBeta2, gpuResultBeta3, gpubeta);
        }

        for(i= 0; i < FEAT_LEN; i++)
        {
            reduceBetas<<< dim3(1), THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double) >>>(numBlocks2 , 1, i, gpuResultBeta3, gpuResultBeta2, gpubeta);
        }


        hipMemcpy(beta, gpubeta, FEAT_LEN * sizeof(double), hipMemcpyDeviceToHost);
        predict_gpu(features,beta,num_features,cpulabels);
        p_corr[i] = calcPCorrect(labels, cpulabels, num_features);
        for(int j = 0; j < FEAT_LEN; j++) {

            beta_array[i*FEAT_LEN +j] = beta[j];
        }
    }

    //choose best beta
    bool break_outer =false;
    for(float f = 0.99 ; f > 0.5 ; f -=0.02)
    {
        for(int i = 0; i< EPOCHS; i++)
        {
            if(p_corr[i].first >= f && p_corr[i].second >= f)
            {
                for(int k = 0; k < FEAT_LEN; k++)
                {
                    beta[k] = beta_array[i*FEAT_LEN +k];
                }
                break_outer = true;
            }
        }
        if(break_outer)
        {
            break;
        }
    }

    hipFree(gpufeatures);
    hipFree(gpulabels);
    hipFree(gpubeta);
    hipFree(gpuResultBeta1);
    hipFree(gpuResultBeta2);
    hipFree(gpuResultBeta3);

    hipHostFree(cpulabels);

    return 0;
}

__global__ void predictKernel(float* features, double* beta, int num_features, int* predictions)
{
    int x = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    double z, proby;
    int pos;

    if (x  >= num_features) {
        return;
    }

    pos = x * FEAT_LEN ;

    for(int j = 0; j < FEAT_LEN; j++) {
            z += beta[j] * features[pos + j];
    }

    proby = (1.0 / (1.0 + exp(-z)));

    if(proby > 0.5)
    {
        predictions[x] = 1;
    }
    else
    {
        predictions[x] = 0;
    }

}

//features, beta and num_features are the input variables, predictions is the output variable
int predict_gpu(float* features, double* beta, int num_features, int* predictions)
{
    float* gpufeatures;
    double* gpubeta;
    int* gpupredictions;

    hipMalloc((void**) &gpufeatures, num_features * FEAT_LEN * sizeof(float));
    hipMemcpy(gpufeatures, features, num_features * FEAT_LEN * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**) &gpubeta, FEAT_LEN * sizeof(double));
    hipMemcpy(gpubeta, beta,     FEAT_LEN * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void**) &gpupredictions, num_features * sizeof(int));
    hipMemset(gpupredictions, 0,        num_features * sizeof(int));


    unsigned int numBlocks = num_features / THREADS_PER_BLOCK + 1 ;


    predictKernel<<< numBlocks, THREADS_PER_BLOCK >>>(gpufeatures, gpubeta, num_features, gpupredictions);

    hipDeviceSynchronize();

    hipMemcpy(predictions, gpupredictions, num_features * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpufeatures);
    hipFree(gpubeta);
    hipFree(gpupredictions);

    return 0;

}


std::pair<float,float> calcPCorrect(int* labels, int* predicted, int num_features)
{
    float total_zeros = 0;
    float total_ones = 0;
    float correct_zeros = 0;
    float correct_ones = 0;

    for(int i = 0; i < num_features; i++) {
        if(labels[i] == predicted[i]) {
            if(labels[i] == 1)
            {
                correct_ones++;
            }
            else
            {
                correct_zeros++;
            }
        }
        if(labels[i] == 1)
        {
            total_ones++;
        }
        else
        {
            total_zeros++;
        }
    }

    return (std::make_pair((correct_ones/total_ones),(correct_zeros/ total_zeros)));
}


/*
int main(int argc, char *argv[])
{
    int num_features = 2;
    float* features = new float[18];
    features[0] = -0.215425;
    features[1]= -0.489831;
    features[2] = -0.733932;
    features[3] = -0.105368;
    features[4] = -0.197275;
    features[5] = -0.288388;
    features[6] = -0.370837;
    features[7] = -0.474699;
    features[8] = -0.538087;
    features[9] = -0.224642;
    features[10] = -0.49868;
    features[11] = -0.743322;
    features[12] = -0.106715;
    features[13] = -0.198606;
    features[14] = -0.29024;
    features[15] = -0.372176;
    features[16] = -0.476292;
    features[17] = -0.539666;
    int* labels = new int[2];
    labels[0] = 0;
    labels[1] = 0;


    double* beta = new double[9];
    train_gpu( labels, features, num_features, beta);
    //features =

}*/
